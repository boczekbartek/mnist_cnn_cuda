#define N 1
#define C 1
#define H 1
#define W 28*28
#define CLASS_NUMBER 10
using namespace std;


vector<hipdnnTensorDescriptor_t*> descriptions;
vector<float*> in_datas;
vector<hipdnnActivationDescriptor_t*> activations;

void add_dense_layer(int batch_size, int feature_of_maps,
                     int height, int width){

    hipdnnTensorDescriptor_t* in_desc = new hipdnnTensorDescriptor_t;
    descriptions.push_back(in_desc);
    CUDNN_CALL(hipdnnCreateTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(*in_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, N, C, H, W));
    float *in_data;
    in_datas.push_back(in_data);
    CUDA_CALL(hipMalloc(in_data, batch_size * feature_of_maps *
                         height * width * sizeof(float)));
}

void add_activation(hipdnnActivationMode_t activation_mode){
  hipdnnActivationDescriptor_t *activation = new hipdnnActivationDescriptor_t;
  activations.push_back(activation);
  checkCUDNN(hipdnnCreateActivationDescriptor(activation));
  checkCUDNN(hipdnnSetActivationDescriptor(activation, activation_mode,
    HIPDNN_PROPAGATE_NAN, 0.0));
}

void add_softmax_activation(){

}

void free_resources(){
  for(int i = 0; i<descriptions.size(); ++i){
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(*descriptions[i]));
    CUDA_CALL(hipFree(*in_datas[i]));
    checkCUDNN(hipdnnDestroyActivationDescriptor(*activations[i]));
  }
}

int main(){
  //TODO: Reading data

  hipdnnHandle_t cudnn;
  CUDNN_CALL(hipdnnCreate(&cudnn));
  add_dense_layer(N, C, H, W);
  add_activation(HIPDNN_ACTIVATION_TANH);
  add_dense_layer(1, 1, 1, CLASS_NUMBER);

  hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_CHANNEL,
                      *descriptions[0], *in_datas[0], //TODO: Check this, datas pointers should be device pointers
                      *descriptions[1], *in_datas[1]);
  //TODO: Training

  free_resources();
  return 0;
}
